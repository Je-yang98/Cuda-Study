#include "hip/hip_runtime.h"
__inline__ __device__ void conservativeCirclesCheck
(float minX, float maxX, float minY, float maxY, size_t circleIdx, size_t tIdx, size_t numCircles, uint* inSection) {
    if (circleIdx >= numCircles){
        inSection[tIdx] = 0;
    } 
    else {
        float3 pos = *(float3*)(&cuConstRendererParams.position[circleIdx * 3]);
        float rad = cuConstRendererParams.radius[circleIdx];

        inSection[tIdx] = static_cast<uint> (circleInBoxConservative(pos.x, pos.y, rad, minX, maxX, maxY, minY));
    }
}


__inline__ __device__ void conservativeCirclesRender
(size_t tIdx, size_t circleIdx, uint* inclusiveOutput, uint* probableCircles) {
    if (tIdx == 0){
        if (inclusiveOutput[0] == 1){ 
            probableCircles[0] = circleIdx;
        }    
    } 
    else if (inclusiveOutput[tIdx] == (inclusiveOutput[tIdx-1]+1)){
        probableCircles[inclusiveOutput[tIdx-1]] = circleIdx;
    }
}


__inline__ __device__ void definiteCirclesCheck
(float minX, float maxX, float minY, float maxY, uint circleIdx, size_t tIdx, uint* inSection) {
    float3 pos = *(float3*)(&cuConstRendererParams.position[circleIdx * 3]);
    float rad = cuConstRendererParams.radius[circleIdx];
    inSection[tIdx] = static_cast<uint> (circleInBox(pos.x, pos.y, rad, minX, maxX, maxY, minY));
}


__inline__ __device__ void definiteCirclesRender
(size_t tIdx, uint* inclusiveOutput, uint* definiteCircles, uint* probableCircles) {
    if (tIdx == 0){
        if (inclusiveOutput[0] == 1){
            definiteCircles[0] = probableCircles[0];
        }
    } 
    else if (inclusiveOutput[tIdx] == (inclusiveOutput[tIdx-1]+1)){
        definiteCircles[inclusiveOutput[tIdx-1]] = probableCircles[tIdx];
    }
}


__inline__ __device__ void sharedMemInclusiveScan
(int threadIndex, uint* sInput, uint* sOutput, volatile uint* sScratch, uint size) {
    if (size > WARP_SIZE) {
        uint idata = sInput[threadIndex];
        uint warpResult = warpScanInclusive(threadIndex, idata, sScratch, WARP_SIZE);
        __syncthreads();

        if ((threadIndex & (WARP_SIZE - 1)) == (WARP_SIZE - 1)){
            sScratch[threadIndex >> LOG2_WARP_SIZE] = warpResult;
        }
        __syncthreads();

        if (threadIndex < (SCAN_BLOCK_DIM / WARP_SIZE)) {
            uint val = sScratch[threadIndex];
            sScratch[threadIndex] = warpScanExclusive(threadIndex, val, sScratch, size >> LOG2_WARP_SIZE);
        }
        __syncthreads();

        sOutput[threadIndex] = warpResult + sScratch[threadIndex >> LOG2_WARP_SIZE];

    } 
    else if (threadIndex < WARP_SIZE) {
        uint idata = sInput[threadIndex];
        sOutput[threadIndex] = warpScanInclusive(threadIndex, idata, sScratch, size);
    }
}

__global__ void kernelRenderCircles() {
    size_t tIdx = blockDim.x * threadIdx.y + threadIdx.x;
    int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelY = blockIdx.y * blockDim.y + threadIdx.y;

    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    float minX = static_cast<float>(blockIdx.x) / gridDim.x;
    float maxX = minX + static_cast<float>(blockDim.x) / imageWidth;
    float minY = static_cast<float>(blockIdx.y) / gridDim.y;
    float maxY = minY + static_cast<float>(blockDim.y) / imageHeight;

    __shared__ uint inSection[BLOCKSIZE];
    __shared__ uint inclusiveOutput[BLOCKSIZE];
    __shared__ uint probableCircles[BLOCKSIZE];
    __shared__ uint scratchPad[2*BLOCKSIZE];
    
    float4* imgPtr;
    float4 color;
    float2 pixelCenterNorm;

    if (pixelX < imageWidth && pixelY < imageHeight) {
        imgPtr = (float4*) &cuConstRendererParams.imageData[4 * (pixelY * imageWidth + pixelX)];
        color = *imgPtr;
        pixelCenterNorm = make_float2(invWidth*(static_cast<float>(pixelX) + 0.5f), invHeight*(static_cast<float>(pixelY) + 0.5f));
    }

    const size_t numCircles = cuConstRendererParams.numCircles;
    
    for (size_t circleIdxStart = 0; circleIdxStart < numCircles; circleIdxStart += BLOCKSIZE) {
        size_t circleIdx = circleIdxStart + tIdx;

        // Find the all circles in the section
        conservativeCirclesCheck(minX, maxX, minY, maxY, circleIdx, tIdx, numCircles, inSection);
        __syncthreads();
        sharedMemInclusiveScan(tIdx, inSection, inclusiveOutput, scratchPad, BLOCKSIZE);
        __syncthreads();
        conservativeCirclesRender(tIdx, circleIdx, inclusiveOutput, probableCircles);
        __syncthreads();

        size_t numConservativeCircles = inclusiveOutput[BLOCKSIZE-1];

        // Find the final circles in the section
        if (tIdx < numConservativeCircles) {
            definiteCirclesCheck(minX, maxX, minY, maxY, probableCircles[tIdx], tIdx, inSection);
        }
        else {
            inSection[tIdx] = 0;
        }
        __syncthreads();
        sharedMemInclusiveScan(tIdx, inSection, inclusiveOutput, scratchPad, BLOCKSIZE);
        __syncthreads();
        uint* definiteCircles = inSection;
        definiteCirclesRender(tIdx, inclusiveOutput, definiteCircles, probableCircles);
        __syncthreads();

        size_t numDefiniteCircles = inclusiveOutput[numConservativeCircles-1];

        // check if pixel is within image
        if (pixelX < imageWidth && pixelY < imageHeight) {
            for (size_t i=0; i<numDefiniteCircles; i++) {
                size_t circleIdx = definiteCircles[i];
                float3 pos = *(float3*)(&cuConstRendererParams.position[circleIdx * 3]);
                shadePixel(circleIdx, pixelCenterNorm, pos, &color);
            }
        }
        __syncthreads();
    }
    if (pixelX < imageWidth && pixelY < imageHeight) {
        *imgPtr = color;
    }

}


__global__ void kernelRenderOneCircle
(short screenMinX, short screenMaxX, short screenMinY, short screenMaxY, float invWidth, float invHeight, int circleIdx){
    int tIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int totalPixel = (screenMaxX - screenMinX) * (screenMaxY - screenMinY);

    if (tIndex <= totalPixel){
        int dimX = screenMaxX - screenMinX;
        int pixelX = tIndex % dimX + screenMinX;
        int pixelY = tIndex / dimX + screenMinY;

        short imageWidth = cuConstRendererParams.imageWidth;

        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4*(imageWidth*pixelY + pixelX)]);
        float2 pixelCenterNorm = make_float2(invWidth*(static_cast<float>(pixelX) + 0.5f), invHeight*(static_cast<float>(pixelY) + 0.5f));
        float3 pos = *(float3*)(&cuConstRendererParams.position[circleIdx*3]);
        shadePixel(circleIdx, pixelCenterNorm, pos, imgPtr);
    }
}


void
CudaRenderer::render() {
    dim3 blockDim(BLOCKDIM, BLOCKDIM);
    size_t gridDimX = (image->width + blockDim.x - 1) / blockDim.x;
    size_t gridDimY = (image->height + blockDim.y - 1) / blockDim.y;
    dim3 gridDim(gridDimX, gridDimY);

    if (numCircles < 7){
        int imageWidth = image->width;
        int imageHeight = image->height;
        float invWidth = 1.f / imageWidth;
        float invHeight = 1.f / imageHeight;
        
        for (int i=0; i<numCircles; i++){
            float px = position[3*i];
            float py = position[3*i + 1];
            float rad = radius[i];

            short minX = static_cast<short>(imageWidth * (px - rad));
            short maxX = static_cast<short>(imageWidth * (px + rad)) + 1;
            short minY = static_cast<short>(imageHeight * (py - rad));
            short maxY = static_cast<short>(imageHeight * (py + rad)) + 1;

            short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
            short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
            short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
            short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;
            
            int totalPixel = (screenMaxX-screenMinX) * (screenMaxY-screenMinY);
            int THREADS_PER_BLOCK = 64;
            int num_blocks = (totalPixel + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            kernelRenderOneCircle<<<num_blocks, THREADS_PER_BLOCK>>>(screenMinX, screenMaxX, screenMinY, screenMaxY, invWidth, invHeight, i);
            hipDeviceSynchronize();
        }
    }
    else {
        kernelRenderCircles<<<gridDim, blockDim>>>();
    } 
    hipDeviceSynchronize();

}
